#include<iostream>
#include<stdlib.h>
#include<time.h>
#include<math.h>
#include <cmath>
#include <hip/hip_runtime.h>
using namespace std;
#define ll long long int

const int Block_Size = 1024;

// This GPU kernel does blockwise in-place scan 
__global__ void Blelloch_Exclusive_Scan(ll *d_in, ll* d_out)
{
    __shared__ ll sh_array[Block_Size];

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    // Copying data from global to shared memory
    sh_array[tid] = d_in[id];
    __syncthreads();

    /** Performing block-wise in-place Blelloch scan **/

    // First step of Blelloch scan : REDUCTION
    for(int k=2; k <= Block_Size; k *= 2)
    {
        if((tid+1) % k == 0)
        {
            sh_array[tid] += sh_array[tid - (k/2)];
        }
        __syncthreads();
    }

    // At the end of reduction, the last element of each block conatins the sum of all elements in that block
    // We store these block-wise sums in d_out
    if(tid == (Block_Size - 1))
    {
        d_out[bid] = sh_array[tid];
        sh_array[tid] = 0;
    }
    __syncthreads();

    // Second step of Blelloch scan : DOWNSWEEP 
    // This is structurally the exact reverse of the reduction step
    for(int k = Block_Size; k >= 2; k /= 2)
    {
        if((tid+1) % k == 0)
        {
            ll temp = sh_array[tid - (k/2)];
            sh_array[tid - (k/2)] = sh_array[tid];
            sh_array[tid] += temp;
        }
        __syncthreads();
    }

    // Copying the scan result back into global memory
    d_in[id] = sh_array[tid];
    // d_in now contains blockwise scan result
    __syncthreads();
}

// This GPU kernel adds the value d_out[id] to all values in the (id)th block of d_in
__global__ void Add(ll* d_in, ll* d_out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;

    d_in[id] += d_out[bid];

    __syncthreads();
}

int main()
{
    ll *h_in, *h_scan;

    int Size;
    cout << "Enter size of the array.\n";
    cin >> Size;
    
    int Reduced_Size = (int)ceil(1.0*Size/Block_Size);
    int Array_Bytes = Size * sizeof(ll);
    int Reduced_Array_Bytes = Reduced_Size * sizeof(ll);
    h_in = (ll*)malloc(Array_Bytes);
    h_scan = (ll*)malloc(Array_Bytes);

    // Populating array with random numbers
    srand(time(0));
    for(ll i=0; i<Size; i++)
    {
        h_in[i] = rand()%10;
    }

    /*cout << "Input Array : \n";
    for(ll i=0; i<Size; i++)
        cout << h_in[i] << " ";
    cout << endl;*/

    ll *d_in, *d_out, *d_sum;

    // GPU Memory allocations
    hipMalloc((void**)&d_in, Reduced_Size*Block_Size*sizeof(ll));  
    // Padding the input array to the next multiple of Block_Size. 
    // The scan algorithm is not dependent on elements past the end of the array, so we don't have to use a special case for the last block.
    hipMalloc((void**)&d_out, Reduced_Array_Bytes);
    hipMalloc((void**)&d_sum, sizeof(ll));

    // Copying input array from CPU to GPU
    hipMemcpy(d_in, h_in, Array_Bytes, hipMemcpyHostToDevice);

    Blelloch_Exclusive_Scan <<< Reduced_Size, Block_Size >>> (d_in, d_out);
    // After first kernel call, d_in has the blockwise scan results and d_out is an auxiliary array that has the blockwise sums

    // Second kernel call is done to scan the blockwise sums array 
    // Then the ith value in the resultant scanned blockwise sums array is added to every value in the ith block
    // This addition step is done in the Add() kernel
    // This is required only if size of the array is greater than the block size
    if(Size > Block_Size)
    {
        Blelloch_Exclusive_Scan <<< 1, Block_Size >>> (d_out, d_sum);
        Add <<< Reduced_Size, Block_Size >>> (d_in, d_out);
    }

    // Copying the result back to the CPU
    hipMemcpy(h_scan, d_in, Array_Bytes, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);

    /*cout << "Exclusive Scan Array : \n";
    for(ll i=0; i<Size; i++)
        cout << h_scan[i] << " ";
    cout << endl;*/

    // CPU computation of the scan
    ll *pref;
    pref = (ll*)malloc(Array_Bytes);
    pref[0] = 0;
    for(ll i=1; i<Size; i++)
        pref[i] = pref[i-1] + h_in[i-1];

    // Checking correctness of the result by comparing CPU and GPU results
    ll flag = 0;
    for(ll i=0; i<Size; i++)
    {
        if(h_scan[i] != pref[i])
        {
            flag = 1;
            break;
        }
    }
    if(flag == 0)
        cout << "Result computed correctly!\n";
    else
        cout << "Result wrong!\n";
}