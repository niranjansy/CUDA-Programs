
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void Array_Add(float* d_A, float* d_B, float* d_Sum)
{
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    d_Sum[id] = d_A[id] + d_B[id];
}

int main()
{
    const int Array_Size = 320000;
    const int Array_Bytes = Array_Size * sizeof(float);
    float h_A[Array_Size], h_B[Array_Size], h_Sum[Array_Size];
    for(int i=0; i<Array_Size; i++)
    {
        h_A[i] = (float)i;
        h_B[i] = (float)i;
    }
    float *d_A, *d_B, *d_Sum;

    //Measuring performance
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc((void**)&d_A, Array_Bytes);
    hipMalloc((void**)&d_B, Array_Bytes);
    hipMalloc((void**)&d_Sum, Array_Bytes);

    hipMemcpy(d_A, h_A, Array_Bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, Array_Bytes, hipMemcpyHostToDevice);

    //Start of performance measurement
    hipEventRecord(start);

    Array_Add<<<625, 512>>>(d_A, d_B, d_Sum);

    //End of performance measurement
    hipEventRecord(stop);

    //Block CPU execution until the event "stop" is recorded
    hipEventSynchronize(stop);

    //Print the time taken in milliseconds
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "The total time taken is " <<  milliseconds << " milliseconds.\n";

    hipMemcpy(h_Sum, d_Sum, Array_Bytes, hipMemcpyDeviceToHost);

    for(int i=0; i<Array_Size; i++)
        cout << h_Sum[i] << " ";

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_Sum);
}