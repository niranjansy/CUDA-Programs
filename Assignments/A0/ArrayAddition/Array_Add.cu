
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void Array_Add(float* d_A, float* d_B, float* d_Sum)
{
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    d_Sum[id] = d_A[id] + d_B[id];
}

int main()
{
    const int Array_Size = 320000;
    const int Array_Bytes = Array_Size * sizeof(float);
    float h_A[Array_Size], h_B[Array_Size], h_Sum[Array_Size];
    for(int i=0; i<Array_Size; i++)
    {
        h_A[i] = (float)i;
        h_B[i] = (float)i;
    }
    float *d_A, *d_B, *d_Sum;

    hipMalloc((void**)&d_A, Array_Bytes);
    hipMalloc((void**)&d_B, Array_Bytes);
    hipMalloc((void**)&d_Sum, Array_Bytes);

    hipMemcpy(d_A, h_A, Array_Bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, Array_Bytes, hipMemcpyHostToDevice);

    Array_Add<<<625, 512>>>(d_A, d_B, d_Sum);

    hipMemcpy(h_Sum, d_Sum, Array_Bytes, hipMemcpyDeviceToHost);

    for(int i=0; i<Array_Size; i++)
        cout << h_Sum[i] << " ";

    hipFree(h_A);
    hipFree(h_B);
    hipFree(h_Sum);
}