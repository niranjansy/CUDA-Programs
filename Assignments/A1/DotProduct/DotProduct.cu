
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

#include <time.h>

__global__ void Array_Add(float* d_out, float* d_array, float Size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    extern __shared__ float sh_array[];
    
    if(id < Size)
        sh_array[tid] = d_array[id];
    __syncthreads();

    for(int s = 512; s>0; s = s/2)
    {
        __syncthreads();
        if(id>=Size || id+s>=Size)
            continue;
        if(tid<s)
            sh_array[tid] += sh_array[tid + s];
    }
    if(tid==0)
        d_out[bid] = sh_array[tid];  

    __syncthreads();
}

float Find_Sum_GPU(float h_array[], int Size)
{

    float* d_array, *d_out, *d_sum;

    hipMalloc((void**)&d_array, Size*sizeof(float));
    hipMalloc((void**)&d_out, ceil(Size*1.0/1024)*sizeof(float));
    hipMalloc((void**)&d_sum, sizeof(float));

    hipMemcpy(d_array, h_array, sizeof(float) * Size, hipMemcpyHostToDevice);

    float h_sum;

    Array_Add <<<ceil(Size*1.0/1024), 1024, 1024*sizeof(float)>>> (d_out, d_array, Size);

    Array_Add <<<1, 1024, 1024*sizeof(float)>>> (d_sum, d_out, ceil(Size*1.0/1024));

    hipMemcpy(&h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_array);
    hipFree(d_out);
    hipFree(d_sum);

    return h_sum;
}

__global__ void Dot_Product(float* d_A, float* d_B, float* d_Prod, int Size)
{
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(id<Size)
        d_Prod[id] = d_A[id] * d_B[id];
}

int main()
{
    int Array_Size;
    cout << "Enetr the size of the two arrays.\n";
    cin >> Array_Size;
    int Array_Bytes = Array_Size * sizeof(float);
    float h_A[Array_Size], h_B[Array_Size], h_Prod[Array_Size];
    for(int i=0; i<Array_Size; i++)
    {
        h_A[i] = (float)i;
        h_B[i] = (float)i;
    }
    float *d_A, *d_B, *d_Prod;

    hipMalloc((void**)&d_A, Array_Bytes);
    hipMalloc((void**)&d_B, Array_Bytes);
    hipMalloc((void**)&d_Prod, Array_Bytes);

    hipMemcpy(d_A, h_A, Array_Bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, Array_Bytes, hipMemcpyHostToDevice);

    Dot_Product<<<ceil(Array_Size*1.0/1024), 1024>>>(d_A, d_B, d_Prod, Array_Size);

    hipMemcpy(h_Prod, d_Prod, Array_Bytes, hipMemcpyDeviceToHost);

    /*for(int i=0; i<Array_Size; i++)
        cout << h_Prod[i] << " ";*/
    
    float Dot_Prod = Find_Sum_GPU(h_Prod, Array_Size);

    cout << "\nThe dot product is " << Dot_Prod << endl;

    hipFree(h_A);
    hipFree(h_B);
    hipFree(h_Prod);
}