#include<iostream>
#include<stdlib.h>
#include<time.h>
#include<math.h>
#include <cmath>
#include <hip/hip_runtime.h>
using namespace std;
#define ll long long int

const int Block_Size = 1024;

// This GPU kernel does blockwise in-place scan 
__global__ void Inclusive_Scan(ll *d_in, ll* d_out)
{
    __shared__ ll sh_array[Block_Size];

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    // Copying data from global to shared memory
    sh_array[tid] = d_in[id];

    __syncthreads();

    // Performing block-wise in-place Hillis-Steele scan
    // In the ith step (starting from 0), to every element we add the element that is (2^i) places to its left, or retain the same element if it does not have so many places to its left
    for(int step = 1; step <= Block_Size; step *= 2)
    {
        if(tid >= step)
        {
            ll temp = sh_array[tid-step];
            __syncthreads();
            sh_array[tid] += temp;
        }
        __syncthreads();
    }
    __syncthreads();

    // Copying the scan result back into global memory
    d_in[id] = sh_array[tid];
    // d_in now contains blockwise scan result
    __syncthreads();

    // Storing the blockwise sums into d_out, i.e, we store the last element in each block of scanned array in its corresponding position in d_out
    if(tid == (Block_Size - 1))
        d_out[bid] = d_in[id];

    __syncthreads();
}

// This GPU kernel adds the value d_out[id] to all values in the (id+1)th block of d_in
__global__ void Add(ll* d_in, ll* d_out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;

    if(bid > 0)
        d_in[id] += d_out[bid-1];

    __syncthreads();
}

int main()
{
    ll *h_in, *h_scan;

    int Size;
    cout << "Enter size of the array.\n";
    cin >> Size;
    
    int Reduced_Size = (int)ceil(1.0*Size/Block_Size);  // The number of blocks that we need to launch
    int Array_Bytes = Size * sizeof(ll);
    int Reduced_Array_Bytes = Reduced_Size * sizeof(ll);

    h_in = (ll*)malloc(Array_Bytes);
    h_scan = (ll*)malloc(Array_Bytes);

    // Populating array with random numbers
    srand(time(0));
    for(ll i=0; i<Size; i++)
    {
        h_in[i] = rand()%10;
    }

    /*cout << "Input Array : \n";
    for(ll i=0; i<Size; i++)
        cout << h_in[i] << " ";
    cout << endl;*/

    ll *d_in, *d_out, *d_sum;

    // GPU Memory allocations
    hipMalloc((void**)&d_in, Reduced_Size*Block_Size*sizeof(ll));  
    // Padding the input array to the next multiple of Block_Size. 
    // The scan algorithm is not dependent on elements past the end of the array, so we don't have to use a special case for the last block.
    hipMalloc((void**)&d_out, Reduced_Array_Bytes);
    hipMalloc((void**)&d_sum, sizeof(ll));

    // Copying input array from CPU to GPU
    hipMemcpy(d_in, h_in, Array_Bytes, hipMemcpyHostToDevice);

    Inclusive_Scan <<< Reduced_Size, Block_Size >>> (d_in, d_out);
    // After first kernel call, d_in has the blockwise scan results and d_out is an auxiliary array that has the blockwise sums

    // Second kernel call is done to scan the blockwise sums array 
    // Then the ith value in the resultant scanned blockwise sums array is added to every value in the (i+1)th block
    // This addition step is done in the Add kernel
    // This is required only if size of the array is greater than the block size
    if(Size > Block_Size)
    {
        Inclusive_Scan <<< 1, Block_Size>>> (d_out, d_sum);
        Add <<< Reduced_Size, Block_Size >>> (d_in, d_out);
    }

    // Copying the result back to the CPU
    hipMemcpy(h_scan, d_in, Array_Bytes, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);

    /*cout << "Inclusive Scan Array : \n";
    for(ll i=0; i<Size; i++)
        cout << h_scan[i] << " ";
    cout << endl;*/

    // CPU computation of the scan
    ll *pref;
    pref = (ll*)malloc(Array_Bytes);
    pref[0] = h_in[0];
    for(ll i=1; i<Size; i++)
        pref[i] = pref[i-1] + h_in[i];

    // Checking correctness of the result by comparing CPU and GPU results
    ll flag = 0;
    for(ll i=0; i<Size; i++)
    {
        if(h_scan[i] != pref[i])
        {
            flag = 1;
            break;
        }
    }
    if(flag == 0)
        cout << "Result computed correctly!\n";
    else
        cout << "Result wrong!\n";
}